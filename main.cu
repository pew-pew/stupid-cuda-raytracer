#include "hip/hip_runtime.h"
#include <iostream>
#include <SDL.h>
#include <GL/glew.h>
#include <exception>
#include <chrono>
#include <tuple>
#include <complex>
#include <iomanip>
#include <cassert>
#include <cuda_gl_interop.h>
#include <surface_functions.h>
#include "geom.h"
#include "orbit.h"

using namespace std::chrono_literals;

void printVal(GLenum tp, std::string name) {
  const GLubyte* sv = glGetString(tp);
  if (sv == nullptr) {
    std::cerr << "can't get " << name << ": " << glewGetErrorString(glGetError()) << std::endl;
  } else {
    std::cerr << name << ": " << reinterpret_cast<const char*>(sv) << std::endl;
  }
}

class SDLError : public std::runtime_error {
  using std::runtime_error::runtime_error;
};

struct SDLOpenGLContext {
  SDL_Window *win = nullptr;
  SDL_GLContext ctx = nullptr;

  SDLOpenGLContext(const std::string& title, int x, int y, int w, int h, uint32_t flags) {
    if (SDL_Init(SDL_INIT_VIDEO) != 0) {
      throw SDLError(std::string("SDL_Init: ") + SDL_GetError());
    }

    assert(0 == SDL_GL_SetAttribute(SDL_GL_CONTEXT_PROFILE_MASK, SDL_GL_CONTEXT_PROFILE_CORE));
    assert(0 == SDL_GL_SetAttribute(SDL_GL_CONTEXT_MAJOR_VERSION, 4));
    assert(0 == SDL_GL_SetAttribute(SDL_GL_CONTEXT_MINOR_VERSION, 6));

    win = SDL_CreateWindow(title.c_str(), x, y, w, h, flags);
    if (!win) {
      SDL_Quit();
      throw SDLError(std::string("SDL_CreateWindow: ") + SDL_GetError());
    }

    ctx = SDL_GL_CreateContext(win);
    if (ctx == nullptr) {
      SDL_DestroyWindow(win);
      SDL_Quit();
      throw SDLError(std::string("SDL_GL_CreateContext: ") + SDL_GetError());
    }

    // ////... glewExperimental
    GLenum glewError = glewInit();
    if (glewError != GLEW_OK) {
      SDL_GL_DeleteContext(ctx);
      SDL_DestroyWindow(win);
      SDL_Quit();
      throw std::runtime_error(
          std::string("glewInit: ")
          + reinterpret_cast<const char *>(glewGetErrorString(glewError))
      );
    }

    // maybe vsync
  }

  ~SDLOpenGLContext() {
    SDL_GL_DeleteContext(ctx);
    SDL_DestroyWindow(win);
    SDL_Quit();
  }
};


__host__ __device__
float mod(float x, float y) {
  return x - std::floor(x / y) * y;
}

using color = std::tuple<int, int, int>;

__host__ __device__
color hsl2rgb(float h, float s, float l) {
  h = mod(h, 360);
  float c = (1 - std::abs(2 * l - 1)) * s;
  float x = c * (1 - std::abs(mod(h / 60, 2) - 1));
  float m = l - c / 2;
  float r_ = 0, g_ = 0, b_ = 0;
  if (h < 60) {
    r_ = c; g_ = x;
  } else if (h < 120) {
    r_ = x; g_ = c;
  } else if (h < 180) {
    g_ = c; b_ = x;
  } else if (h < 240) {
    g_ = x; b_ = c;
  } else if (h < 300) {
    r_ = x; b_ = c;
  } else {
    r_ = c; b_ = x;
  }
  return {(r_ + m) * 255, (g_ + m) * 255, (b_ + m) * 255};
}


struct World {
  float pos_x = 0, pos_y = 0;

  __device__
  color worldAt(float x, float y) {
    if ((Vec(x, y) - Vec(pos_x, pos_y)).lensq() < 0.1 * 0.1) {
      return {0, 60, 20};
    }

    x = std::floor(x * 10) / 10;
    y = std::floor(y * 10) / 10;
    return {
        int(mod(int(x * 255), 255)),
        int(mod(int(y * 255), 255)),
        125,
    };
  }

  __device__
  color viewAt(float dx, float dy, float t) {
    float brightness = 1;

    Vec pos(pos_x, pos_y);
    Vec d(dx, dy);

//    Vec m0(1, 0);
//    Vec d0 = Vec(1, 1).rotateBy(t / 10);
//    float itMy = intersectionTime(pos, d, m0, d0);
//    float itOther = intersectionTime(m0, d0, pos, d);
//    Vec fin;
//    if (itOther < 0 || itOther > 1 || itMy < 0 || itMy > 1) {
//      fin = pos + d;
//    } else {
//      brightness = 0.6;
//      Vec mid = pos + d * itMy;
//      fin = mid + d.symmetryOff(d0) * (1 - itMy);
////      fin = pos + d * itMy;
//    }

    Vec vel = d / d.len();
    float dist = d.len();
    Vec c0(0, 1);

//    Vec fin = c0 + twobody(0.001, dist, (pos - c0), vel);



    float dt = 0.001;
    for (int i = 0; i < 10000 && dist > 0; i++) {
      Vec r = (c0 - pos);
      Vec f = r / (r.len() * r.lensq());
      vel += f * dt;
      pos += vel * dt;
      dist -= vel.len() * dt;
    }
    Vec fin = pos;

    int r, g, b;
    auto col = worldAt(fin.x, fin.y);
    r = std::get<0>(col);
    g = std::get<1>(col);
    b = std::get<2>(col);

//    float k = std::max(0.3f, 1 - (dx*dx + dy*dy));
    return {r * brightness, g * brightness, b * brightness};
  }
};


struct Axis {
  bool neg = false, pos = false;

  float delta() {
    return (neg * -1.0f + pos * 1.0f);
  }
};

template< typename T >
std::string int_to_hex( T i )
{
  std::stringstream stream;
  stream << "0x"
         << std::setfill ('0') << std::setw(sizeof(T)*2)
         << std::hex << i;
  return stream.str();
}

void checkErr(int line_num, std::string line) {
  GLenum err = glGetError();
  if (err != GL_NO_ERROR) {
    std::cerr << line_num << ": " << line << "\ngl error: " << glewGetErrorString(err) << " (" << int_to_hex(err) << ")" << std::endl;
    throw std::runtime_error(std::to_string(err));
  }
}

#define glGuard(expr) \
do { \
  glGetError(); \
  expr; \
  checkErr(__LINE__, #expr); \
} while (false)



__global__
void render(int w, int h, hipSurfaceObject_t surf, World world, float t) {
  float k = std::min(w, h) / 2.0f;

  int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = blockDim.x * gridDim.x;

  int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  int stride_y = blockDim.y * gridDim.y;

  for (int y = idx_y; y < h; y += stride_y) {
    for (int x = idx_x; x < w; x += stride_x) {
      float rel_x = (x + 0.5f - w / 2.0f) / k;
      float rel_y = (y + 0.5f - h / 2.0f) / k;

      auto col = world.viewAt(rel_x, rel_y, t);
      surf2Dwrite(make_uchar4(
          std::get<0>(col),
          std::get<1>(col),
          std::get<2>(col),
          0
      ), surf, x * 4, y);
    }
  }
}


void dow() {
  const int W = 800;
  const int H = 600;
  auto ctx = SDLOpenGLContext("Hello!", 100, 100, W, H, SDL_WINDOW_SHOWN | SDL_WINDOW_OPENGL);

  printVal(GL_RENDERER, "GL_VENDOR");
  printVal(GL_VENDOR, "GL_RENDERER");
  printVal(GL_VERSION, "GL_VERSION");

//  std::cout << "..." << std::endl;
//
//  Vec end = twobody(0.01, 0.01, Vec(0, 100), Vec(0, -1));
//
//  std::cout << end << std::endl;

//  return;

  // https://stackoverflow.com/questions/31482816/opengl-is-there-an-easier-way-to-fill-window-with-a-texture-instead-using-vbo
  GLuint fb = 0;
  glGuard(glGenFramebuffers(1, &fb));
  glGuard(glBindFramebuffer(GL_READ_FRAMEBUFFER, fb));

  GLuint tex = 0;
  glGuard(glGenTextures(1, &tex));
  glGuard(glBindTexture(GL_TEXTURE_2D, tex));

  glGuard(glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, W, H, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr));
  glGuard(glBindTexture(GL_TEXTURE_2D, 0));

  glGuard(glFramebufferTexture2D(GL_READ_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, tex, 0));

  glBindFramebuffer(GL_DRAW_FRAMEBUFFER, 0);

  // https://forums.developer.nvidia.com/t/reading-and-writing-opengl-textures-with-cuda/31746/6
  hipGraphicsResource *resource;
  assert(hipSuccess == hipGraphicsGLRegisterImage(&resource, tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
  assert(hipSuccess == hipGraphicsMapResources(1, &resource));

  hipArray_t writeArray;
  assert(hipSuccess == hipGraphicsSubResourceGetMappedArray(&writeArray, resource, 0, 0));

  hipResourceDesc descr = {};
  descr.resType = hipResourceTypeArray;
  descr.res.array.array = writeArray;

  hipSurfaceObject_t surf;
  assert(hipSuccess == hipCreateSurfaceObject(&surf, &descr));

  Axis dx, dy;
  World world;

  auto prev_frame = std::chrono::steady_clock::now();
  auto start = prev_frame;
  while (true) {
    auto now = std::chrono::steady_clock::now();
    float dt = std::chrono::duration_cast<std::chrono::duration<float, std::chrono::seconds::period>>(now - prev_frame).count();
    prev_frame = now;

    std::cout << dt - 1/60.0f << std::endl;

    SDL_Event evt;
    bool quit = false;
    while (SDL_PollEvent(&evt)) {
      if (evt.type == SDL_QUIT) {
        quit = true;
      }
      if (evt.type == SDL_KEYDOWN) {
        switch (evt.key.keysym.sym) {
          case (SDLK_d):
            dx.pos = true; break;
          case (SDLK_a):
            dx.neg = true; break;
          case (SDLK_w):
            dy.pos = true; break;
          case (SDLK_s):
            dy.neg = true; break;
        }
      }
      if (evt.type == SDL_KEYUP) {
        switch (evt.key.keysym.sym) {
          case (SDLK_d):
            dx.pos = false; break;
          case (SDLK_a):
            dx.neg = false; break;
          case (SDLK_w):
            dy.pos = false; break;
          case (SDLK_s):
            dy.neg = false; break;
        }
      }
    }
    if (quit)
      break;

    float t = std::chrono::duration_cast<std::chrono::duration<float, std::chrono::seconds::period>>(prev_frame - start).count();

    world.pos_x += dx.delta() * dt;
    world.pos_y += dy.delta() * dt;

    render<<<10, 256>>>(W, H, surf, world, t);
    hipDeviceSynchronize();

    // TODO: clear?
    glGuard(glBlitFramebuffer(0, 0, W, H, 0, 0, W, H, GL_COLOR_BUFFER_BIT, GL_NEAREST));

    SDL_GL_SwapWindow(ctx.win);

    auto left = std::chrono::steady_clock::now() - prev_frame;
    SDL_Delay(std::max(0.0f, 1000.0f / 60 - std::chrono::duration_cast<std::chrono::milliseconds>(left).count()));
  }

  assert(hipSuccess == hipDestroySurfaceObject(surf));
  assert(hipSuccess == hipGraphicsUnmapResources(1, &resource));
  assert(hipSuccess == hipGraphicsUnregisterResource(resource));
}

int main(int, char**) {
  try {
    dow();
  } catch (const std::exception &e) {
    std::cerr << e.what() << std::endl;
    return 1;
  }
  return 0;
}